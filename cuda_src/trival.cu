#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

int main(int argc, char** argv)
{
  // Host element type definition
  typedef float floatTypeA;
  typedef float floatTypeB;
  typedef float floatTypeC;
  typedef float floatTypeCompute;

  // CUDA types
  hipDataType typeA = HIP_R_32F;
  hipDataType typeB = HIP_R_32F;
  hipDataType typeC = HIP_R_32F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

  floatTypeCompute alpha = (floatTypeCompute)1.1f;
  floatTypeCompute beta  = (floatTypeCompute)0.9f;

  printf("Include headers and define data types\n");

  return 0;
}
