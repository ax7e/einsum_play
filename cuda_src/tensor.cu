#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>

// Handle cuTENSOR errors
#define HANDLE_ERROR(x) {                                                              \
  const auto err = x;                                                                  \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                                 \
  { printf("Error: %s in line %d\n", hiptensorGetErrorString(err), __LINE__); exit(-1); } \
}
using std::vector; 

void generateTestDataDim(int p, int q, int numberOfCell,
    int64_t &I, int64_t &J, int64_t &K, int64_t &L, int64_t &M, int geometryDim = 3, int topologyDim = 3) {
    I = q * (q+1) * (q+2) / 6;
    K = M = (p+1) * (p+2) * (p+3) / 6;
    J = numberOfCell;
    L = geometryDim;
    printf("Data benchmark:[I,J,K,L,M]=[%d,%d,%d,%d,%d]\n", I, J, K, L, M);
}

typedef float floatType;
typedef HIP_R_32F tensorType;
typedef float floatTypeCompute;

void initTenosr(const vector<int> &mode, int &nmode, const vector<int> &extentLib, std::unordered_map<int, int64_t> &extent, size_t &eleCnt, size_t &sizeByte, void *&T_d, floatType *& T_h, 
    hiptensorHandle_t &handle, hiptensorTensorDescriptor_t &desc, uint32_t &align) {
    nmode = mode.size(); 
    for(auto m : mode) extent.push_back(extentLib[m]);
    eleCnt = 1; 
    for(auto m : mode) eleCnt *= extentLib[m];
    sizeByte = sizeof(floatType) * eleCnt;
    hipMalloc((void**)&T_d, sizeByte);
    T_h = (floatType*) malloc(sizeof(floatType) * eleCnt);
    for (int64_t i = 0; i < elementsA; i++)
      T_h[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    hipMemcpy(T_d, T_h, sizeByte, hipMemcpyHostToDevice);
    HANDLE_ERROR( hiptensorInitTensorDescriptor( &handle,
      &desc,
      nmode,
      extent.data(),
      NULL,/*stride*/
      tensorType, HIPTENSOR_OP_IDENTITY ) ); 
    HANDLE_ERROR( cutensorGetAlignmentRequirement( &handle,
       T_d,
       &desc,
       &align) ); 
}

int main(int argc, char** argv)
{
  //e(j,k,m)=a(i)*b(i,j,k,l)*c(i,j,m,l)*d(j)
  //e(j,k,m)=a(i)*f(i,j,k,m,l)*d(j)
  //e(j,k,m)=g(i,j)*f(i,j,k,m,l)

  // CUDA types
  hipDataType type = HIP_R_32F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

  printf("Include headers and define data types\n");

  /* ***************************** */

  // Create vector of modes
  std::vector<int> modeA{'i'}, 
                   modeB{'i','j','k','l'}, 
                   modeC{'i','j','m','l'}, 
                   modeD{'j'}, 
                   modeE{'j','k','m'}, 
                   modeF{'i','j','k','m','l'}, 
                   modeG{'i','j'};
  int nmodeA, nmodeB, nmodeC, nmodeD, nmodeE, nmodeF, nmodeG;

  // Extents
  std::unordered_map<int, int64_t> extent;
  generateTestDataDim(3, 3, 100, extent['i'], extent['j'], extent['k'], extent['m'], extent['l']);

  // Create a vector of extents for each tensor
  std::vector<int64_t> extentG, extentF, extentE, extentD, extentC, extentA, extentB;
  // Number of elements of each tensor
  size_t elementsA,elementsB,elementsC,elementsD,elementsE,elementsF,elementsG;


  // Size in bytes
  size_t sizeA,sizeB,sizeC,sizeD,sizeE,sizeF,sizeG;

  // Allocate on device
  void *A_d, *B_d, *C_d, *D_d, *E_d, *F_d, *G_d;

  // Allocate on host
  floatTypeA *A_h,*B_h,*C_h,*D_h,*E_h,*F_h,*G_h;

  // Initialize cuTENSOR library
  hiptensorHandle_t handle;
  cutensorInit(&handle);

  // Create Tensor Descriptors
  hiptensorTensorDescriptor_t descA,descB,descC,descD,descE,descF,descG;

  uint32_t alignA,alignB,alignC,alignD,alignE,alignF,alignG;
  initTenosr(modeA,nmodeA,extent,extentA,elementsA,sizeA,A_d,A_h,handle,descA,alignA);
  initTenosr(modeB,nmodeB,extent,extentB,elementsB,sizeB,B_d,B_h,handle,descB,alignB);
  initTenosr(modeC,nmodeC,extent,extentC,elementsC,sizeC,C_d,C_h,handle,descC,alignC);
  initTenosr(modeD,nmodeD,extent,extentD,elementsD,sizeD,D_d,D_h,handle,descD,alignD);
  initTenosr(modeE,nmodeE,extent,extentE,elementsE,sizeE,E_d,E_h,handle,descE,alignE);
  initTenosr(modeF,nmodeF,extent,extentF,elementsF,sizeF,F_d,F_h,handle,descF,alignF);
  initTenosr(modeG,nmodeG,extent,extentG,elementsG,sizeG,G_d,G_h,handle,descG,alignG);
  // Create the Contraction Descriptor
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( &handle,
              &desc,
              &descA, modeA.data(), alignmentRequirementA,
              &descB, modeB.data(), alignmentRequirementB,
              &descC, modeC.data(), alignmentRequirementC,
              &descC, modeC.data(), alignmentRequirementC,
              typeCompute) );

  printf("Initialize contraction descriptor\n");

  /* ***************************** */

  // Set the algorithm to use
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
              &handle, &find,
              HIPTENSOR_ALGO_DEFAULT) );

  printf("Initialize settings to find algorithm\n");

  /* ***************************** */

  // Query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(&handle,
              &desc,
              &find,
              CUTENSOR_WORKSPACE_RECOMMENDED, &worksize ) );

  // Allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
      if( hipSuccess != hipMalloc(&work, worksize) ) // This is optional!
      {
          work = nullptr;
          worksize = 0;
      }
  }

  printf("Query recommended workspace size and allocate it\n");

  /* ***************************** */

  // Create Contraction Plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(&handle,
                                            &plan,
                                            &desc,
                                            &find,
                                            worksize) );

  printf("Create plan for contraction\n");

  /* ***************************** */

  hiptensorStatus_t err;

  // Execute the tensor contraction
  err = hiptensorContraction(&handle,
                            &plan,
                     (void*)&alpha, A_d,
                                    B_d,
                     (void*)&beta,  C_d,
                                    C_d,
                            work, worksize, 0 /* stream */);
  hipDeviceSynchronize();

  // Check for errors
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
      printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  printf("Execute contraction from plan\n");

  /* ***************************** */

  if ( A ) free( A );
  if ( B ) free( B );
  if ( C ) free( C );
  if ( A_d ) hipFree( A_d );
  if ( B_d ) hipFree( B_d );
  if ( C_d ) hipFree( C_d );
  if ( work ) hipFree( work );

  printf("Successful completion\n");

  return 0;
}